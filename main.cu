#include "hip/hip_runtime.h"
/*****************************************************************************
 This file is part of the XLQC program.                                      
 Copyright (C) 2015 Xin Li <lixin.reco@gmail.com>                            
                                                                           
 Filename:  main.cu                                                      
 License:   BSD 3-Clause License

 This software is provided by the copyright holders and contributors "as is"
 and any express or implied warranties, including, but not limited to, the
 implied warranties of merchantability and fitness for a particular purpose are
 disclaimed. In no event shall the copyright holder or contributors be liable
 for any direct, indirect, incidental, special, exemplary, or consequential
 damages (including, but not limited to, procurement of substitute goods or
 services; loss of use, data, or profits; or business interruption) however
 caused and on any theory of liability, whether in contract, strict liability,
 or tort (including negligence or otherwise) arising in any way out of the use
 of this software, even if advised of the possibility of such damage.
 *****************************************************************************/

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <string>

#include <gsl/gsl_math.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_eigen.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_linalg.h>

#include "int_lib/cints.h"
#include "int_lib/crys.h"
#include "int_lib/chgp.h"

#include "typedef.h"
#include "basis.h"
#include "scf.h"

#include "cuda_rys.h"

void my_cuda_safe(hipError_t err, std::string word)
{
    if(err != hipSuccess) 
    {
		fprintf(stderr, "Error during %s: ", word.c_str());

		// check for error
		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
			exit(-1);
		}
    } 
}

int main(int argc, char* argv[])
{
	Atom   *p_atom   = (Atom *)my_malloc(sizeof(Atom) * 1);
	Basis  *p_basis  = (Basis *)my_malloc(sizeof(Basis) * 1);

	//====== parse geom.dat ========

	// get number of atoms
	p_atom->num = get_natoms();
	fprintf(stdout, "Natoms = %d\n", p_atom->num);

	// atomic coordinates and atom name
	p_atom->pos = (double **)my_malloc(sizeof(double *) * p_atom->num);
	p_atom->name = (char **)my_malloc(sizeof(char *) * p_atom->num);

	int iatom;
	for (iatom = 0; iatom < p_atom->num; ++ iatom)
	{
		p_atom->pos[iatom] = (double *)my_malloc(sizeof(double) * CART_DIM);
		p_atom->name[iatom] = (char *)my_malloc(sizeof(char) * 5);
	}

	// nuclear charge
	p_atom->nuc_chg = (int *)my_malloc(sizeof(int) * p_atom->num);

	// read atomic positions, nuclear charge and atom name
	read_geom(p_atom);

	fprintf(stdout, "Coordinates in atomic unit:\n");
	for (iatom = 0; iatom < p_atom->num; ++ iatom)
	{
		fprintf(stdout, "%s (%.1f)  %.10f  %.10f  %.10f\n", 
				p_atom->name[iatom], (double)p_atom->nuc_chg[iatom],
				p_atom->pos[iatom][0], p_atom->pos[iatom][1], p_atom->pos[iatom][2]);
	}
	
	// nuclear repulsion energy
	double ene_nucl = calc_ene_nucl(p_atom);
	fprintf(stdout, "Nuclear repulsion = %-20.10f\n", ene_nucl);


	//====== parse basis.dat ========

	// parse basis functions
	parse_basis(p_atom, p_basis);

	fprintf(stdout, "System Nbasis = %d\n", p_basis->num);

	// basis function exponents, coefficients, and normalization factors
	p_basis->expon = (double **)my_malloc(sizeof(double *) * p_basis->num);
	p_basis->coef  = (double **)my_malloc(sizeof(double *) * p_basis->num);
	p_basis->norm  = (double **)my_malloc(sizeof(double *) * p_basis->num);

	// number of primitive functions in each contracted funciton
	p_basis->nprims = (int *)my_malloc(sizeof(int) * p_basis->num);

	// Cartesian coordinates and l,m,n numbers
	p_basis->xbas  = (double **)my_malloc(sizeof(double *) * p_basis->num);
	p_basis->lmn = (int **)my_malloc(sizeof(int *) * p_basis->num);

	int ibasis;
	for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
	{
		p_basis->xbas[ibasis] = (double *)my_malloc(sizeof(double) * CART_DIM);
		p_basis->lmn[ibasis]  = (int *)my_malloc(sizeof(int) * CART_DIM);
	}

	// read basis set (also calculate normalization factors)
	read_basis(p_atom, p_basis);

#ifdef DEBUG
	print_basis(p_basis);
#endif


	//====== one- and two-electron integrals ========

	// overlap, kinetic energy and nuclear attraction integral
	gsl_matrix *S = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_matrix *T = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_matrix *V = gsl_matrix_alloc(p_basis->num, p_basis->num);

	// two-electron ingetral
	int n_combi = p_basis->num * (p_basis->num + 1) / 2;
	int n_eri = n_combi * (n_combi + 1) / 2;
	//fprintf(stdout, "N_eri = %d\n", n_eri);
	//double *ERI = (double *)my_malloc_2(sizeof(double) * n_eri, "ERI");

	int a,b;
	for (a = 0; a < p_basis->num; ++ a)
	{
		for (b = 0; b <= a; ++ b)
		{
			// overlap
			double s = calc_int_overlap(p_basis, a, b);

			// kinetic energy
			double t = calc_int_kinetic(p_basis, a, b);

			// nuclear repulsion
			double v = calc_int_nuc_attr(p_basis, a, b, p_atom);

			// save one-electron integrals in matrices
			gsl_matrix_set(S, a, b, s);
			gsl_matrix_set(T, a, b, t);
			gsl_matrix_set(V, a, b, v);
			if (a != b)
			{
				gsl_matrix_set(S, b, a, s);
				gsl_matrix_set(T, b, a, t);
				gsl_matrix_set(V, b, a, v);
			}

			/*
			// two-electron integral
			int ij = ij2intindex(a, b);
			for (int c = 0; c <= a; ++ c)
			{
				int d_max = (a == c) ? b : c;
				for (int d = 0; d <= d_max; ++ d)
				{
					int kl = ij2intindex(c, d);
					//if (ij < kl) { continue; }

					int ijkl = ij2intindex(ij, kl);

					double eri = calc_int_eri_rys(p_basis, a, b, c, d);

					ERI[ijkl] = eri;
				}
			}
			*/
		}
	}


	// count number of primitive integrals in a <bra| or |ket>
	int count_prim = 0;
	int i,j;
	for (a = 0; a < p_basis->num; ++ a)
	{
		int lena = p_basis->nprims[a];
		for (b = 0; b <= a; ++ b)
		{
			int lenb = p_basis->nprims[b];
        
			for (i=0; i<lena; i++)
				for (j=0; j<lenb; j++)
					++ count_prim;
		}
	}

	// allocate memory for arrays on host
	size_t n_CI_bytes = sizeof(double) * n_combi;
	size_t n_CI_bytes_int = sizeof(int) * n_combi;
	size_t n_PI_bytes = sizeof(double) * count_prim;
	size_t n_ERI_bytes = sizeof(double) * n_eri;

	double *h_xa = (double *)my_malloc(n_CI_bytes);
	double *h_ya = (double *)my_malloc(n_CI_bytes);
	double *h_za = (double *)my_malloc(n_CI_bytes);
	int    *h_la = (int    *)my_malloc(n_CI_bytes_int);
	int    *h_ma = (int    *)my_malloc(n_CI_bytes_int);
	int    *h_na = (int    *)my_malloc(n_CI_bytes_int);
	double *h_aexps = (double *)my_malloc(n_PI_bytes);
	double *h_acoef = (double *)my_malloc(n_PI_bytes);
	// note that 'anorm' is absorbed into 'acoef'

	double *h_xb = (double *)my_malloc(n_CI_bytes);
	double *h_yb = (double *)my_malloc(n_CI_bytes);
	double *h_zb = (double *)my_malloc(n_CI_bytes);
	int    *h_lb = (int    *)my_malloc(n_CI_bytes_int);
	int    *h_mb = (int    *)my_malloc(n_CI_bytes_int);
	int    *h_nb = (int    *)my_malloc(n_CI_bytes_int);
	double *h_bexps = (double *)my_malloc(n_PI_bytes);
	double *h_bcoef = (double *)my_malloc(n_PI_bytes);
	// note that 'bnorm' is absorbed into 'bcoef'

	int *h_start_contr = (int *)my_malloc(n_CI_bytes_int);
	int *h_end_contr   = (int *)my_malloc(n_CI_bytes_int);

	double *h_eri = (double *)my_malloc(n_ERI_bytes);

	// fill arrays on host
	// index counts primitive integrals
	// index_contr counts contracted integrals
	int index = 0;
	int index_contr = 0;

	for (a = 0; a < p_basis->num; ++ a)
	{
		int lena = p_basis->nprims[a];
		for (b = 0; b <= a; ++ b)
		{
			int lenb = p_basis->nprims[b];

			h_start_contr[index_contr] = index;

			h_xa[index_contr] = p_basis->xbas[a][0];
			h_ya[index_contr] = p_basis->xbas[a][1];
			h_za[index_contr] = p_basis->xbas[a][2];

			h_la[index_contr] = p_basis->lmn[a][0];
			h_ma[index_contr] = p_basis->lmn[a][1];
			h_na[index_contr] = p_basis->lmn[a][2];
        
			h_xb[index_contr] = p_basis->xbas[b][0];
			h_yb[index_contr] = p_basis->xbas[b][1];
			h_zb[index_contr] = p_basis->xbas[b][2];
                            
			h_lb[index_contr] = p_basis->lmn[b][0];
			h_mb[index_contr] = p_basis->lmn[b][1];
			h_nb[index_contr] = p_basis->lmn[b][2];

			int i,j;
			for (i=0; i<lena; i++)
			{
				for (j=0; j<lenb; j++)
				{
					h_aexps[index] = p_basis->expon[a][i];
					h_acoef[index] = p_basis->coef[a][i] * p_basis->norm[a][i];
					// note that 'anorm' is absorbed into 'acoef'

					h_bexps[index] = p_basis->expon[b][j];
					h_bcoef[index] = p_basis->coef[b][j] * p_basis->norm[b][j];
					// note that 'bnorm' is absorbed into 'bcoef'

					++ index;
				}
			}

			h_end_contr[index_contr] = index - 1;

			++ index_contr;
		}
	}
	printf("Num_Prim_Combi  = %d (%d)\n", index, count_prim);
	printf("Num_Contr_Combi = %d (%d)\n", index_contr, n_combi);

	// initialize arrays on device
	double *dev_xa, *dev_ya, *dev_za;
	double *dev_xb, *dev_yb, *dev_zb;
	int    *dev_la, *dev_ma, *dev_na;
	int    *dev_lb, *dev_mb, *dev_nb;
	double *dev_aexps, *dev_acoef;
	double *dev_bexps, *dev_bcoef;

	dev_xa = NULL; dev_ya = NULL; dev_za = NULL;
	dev_xb = NULL; dev_yb = NULL; dev_zb = NULL;
	dev_la = NULL; dev_ma = NULL; dev_na = NULL;
	dev_lb = NULL; dev_mb = NULL; dev_nb = NULL;
	dev_aexps = NULL; dev_acoef = NULL;
	dev_bexps = NULL; dev_bcoef = NULL;

	int *dev_start_contr = NULL;
	int *dev_end_contr   = NULL;

	double *dev_eri = NULL;

	// allocate memories for arrays on device
	fprintf(stdout, "Mem_on_Device = %zu MB\n",
			(n_CI_bytes*8 + n_PI_bytes*4 + n_CI_bytes_int*6 + n_ERI_bytes) / 1000000);

	hipMalloc((void**)&dev_xa, n_CI_bytes);
	hipMalloc((void**)&dev_ya, n_CI_bytes);
	hipMalloc((void**)&dev_za, n_CI_bytes);
	hipMalloc((void**)&dev_xb, n_CI_bytes);
	hipMalloc((void**)&dev_yb, n_CI_bytes);
	hipMalloc((void**)&dev_zb, n_CI_bytes);

	if(dev_xa == NULL || dev_ya == NULL || dev_za == NULL ||
	   dev_xb == NULL || dev_yb == NULL || dev_zb == NULL)
	{
		printf("Error: cannot hipMalloc for x_basis!\n");
		exit(1);
	}

	hipMalloc((void**)&dev_la, n_CI_bytes_int);
	hipMalloc((void**)&dev_ma, n_CI_bytes_int);
	hipMalloc((void**)&dev_na, n_CI_bytes_int);
	hipMalloc((void**)&dev_lb, n_CI_bytes_int);
	hipMalloc((void**)&dev_mb, n_CI_bytes_int);
	hipMalloc((void**)&dev_nb, n_CI_bytes_int);

	if(dev_la == NULL || dev_ma == NULL || dev_na == NULL ||
	   dev_lb == NULL || dev_mb == NULL || dev_nb == NULL)
	{
		printf("Error: cannot hipMalloc for l_basis!\n");
		exit(1);
	}

	hipMalloc((void**)&dev_aexps, n_PI_bytes);
	hipMalloc((void**)&dev_acoef, n_PI_bytes);
	hipMalloc((void**)&dev_bexps, n_PI_bytes);
	hipMalloc((void**)&dev_bcoef, n_PI_bytes);

	if(dev_aexps == NULL || dev_acoef == NULL ||
	   dev_bexps == NULL || dev_bcoef == NULL)
	{
		printf("Error: cannot hipMalloc for exp_basis!\n");
		exit(1);
	}

	hipMalloc((void**)&dev_start_contr, n_CI_bytes);
	hipMalloc((void**)&dev_end_contr,   n_CI_bytes);

	hipMalloc((void**)&dev_eri, n_ERI_bytes);

	if(dev_eri == NULL || dev_start_contr == NULL || dev_end_contr == NULL)
	{
		printf("Error: cannot hipMalloc for dev_eri!\n");
		exit(1);
	}

	// copy data from host to device
	my_cuda_safe(hipMemcpy(dev_xa, h_xa, n_CI_bytes, hipMemcpyHostToDevice),"mem_xa");
	my_cuda_safe(hipMemcpy(dev_ya, h_ya, n_CI_bytes, hipMemcpyHostToDevice),"mem_ya");
	my_cuda_safe(hipMemcpy(dev_za, h_za, n_CI_bytes, hipMemcpyHostToDevice),"mem_za");
	my_cuda_safe(hipMemcpy(dev_xb, h_xb, n_CI_bytes, hipMemcpyHostToDevice),"mem_xb");
	my_cuda_safe(hipMemcpy(dev_yb, h_yb, n_CI_bytes, hipMemcpyHostToDevice),"mem_yb");
	my_cuda_safe(hipMemcpy(dev_zb, h_zb, n_CI_bytes, hipMemcpyHostToDevice),"mem_zb");

	my_cuda_safe(hipMemcpy(dev_la, h_la, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_la");
	my_cuda_safe(hipMemcpy(dev_ma, h_ma, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_ma");
	my_cuda_safe(hipMemcpy(dev_na, h_na, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_na");
	my_cuda_safe(hipMemcpy(dev_lb, h_lb, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_lb");
	my_cuda_safe(hipMemcpy(dev_mb, h_mb, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_mb");
	my_cuda_safe(hipMemcpy(dev_nb, h_nb, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_nb");

	my_cuda_safe(hipMemcpy(dev_aexps, h_aexps, n_PI_bytes, hipMemcpyHostToDevice),"mem_ae");
	my_cuda_safe(hipMemcpy(dev_acoef, h_acoef, n_PI_bytes, hipMemcpyHostToDevice),"mem_ac");
	my_cuda_safe(hipMemcpy(dev_bexps, h_bexps, n_PI_bytes, hipMemcpyHostToDevice),"mem_be");
	my_cuda_safe(hipMemcpy(dev_bcoef, h_bcoef, n_PI_bytes, hipMemcpyHostToDevice),"mem_bc");

	my_cuda_safe(hipMemcpy(dev_start_contr, h_start_contr, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_start");
	my_cuda_safe(hipMemcpy(dev_end_contr,   h_end_contr,   n_CI_bytes_int, hipMemcpyHostToDevice),"mem_end");


	// create 8x8 thread blocks
	dim3 block_size;
	block_size.x = 8;
	block_size.y = 8;

	// configure a two dimensional grid as well
	dim3 grid_size;
	grid_size.x = n_combi / block_size.x + (n_combi % block_size.x ? 1 : 0);
	grid_size.y = n_combi / block_size.y + (n_combi % block_size.y ? 1 : 0);


	// launch the kernel to calculate two-electron integrals on GPU
	cuda_rys_eri_2d<<<grid_size, block_size>>>
		(dev_xa,dev_ya,dev_za, dev_la,dev_ma,dev_na, dev_aexps,dev_acoef,
		 dev_xb,dev_yb,dev_zb, dev_lb,dev_mb,dev_nb, dev_bexps,dev_bcoef,
		 n_combi, dev_start_contr, dev_end_contr, dev_eri);

	// copy the results back to host
	my_cuda_safe(hipMemcpy(h_eri, dev_eri, n_ERI_bytes, hipMemcpyDeviceToHost),"mem_eri"); 


	/* just for test...
	int check_passed = 1;
	for (i = 0; i < n_eri; ++ i)
	{
		double diff = fabs(h_eri[i]-ERI[i]);
		if (diff > 1e-12)
		{
			check_passed = 0;
			printf("%-8d %18.12f %18.12f   %18.12f\n", i, ERI[i], h_eri[i], ERI[i]-h_eri[i]);
		}
	}
	if (check_passed) { printf("Check passed!\n"); }
	*/


	//====== start SCF calculation ========

	// NOTE: assume zero charge and closed-shell electronics structure
	int n_elec = 0;
	for (iatom = 0; iatom < p_atom->num; ++ iatom)
	{
		n_elec += p_atom->nuc_chg[iatom];
	}

	if (n_elec % 2 != 0)
	{
		fprintf(stderr, "Error: Number of electrons (%d) is not even!\n", n_elec);
	}

	int n_occ = n_elec / 2;


	// get core Hamiltonian
	gsl_matrix *H_core = gsl_matrix_alloc(p_basis->num, p_basis->num);
	sum_H_core(p_basis->num, H_core, T, V);

	// get S^-1/2
	gsl_matrix *S_invsqrt = gsl_matrix_alloc(p_basis->num, p_basis->num);
	diag_overlap(p_basis->num, S, S_invsqrt);

#ifdef DEBUG
	printf("S:\n"); my_print_matrix(S);
	printf("T:\n"); my_print_matrix(T);
	printf("V:\n"); my_print_matrix(V);
	printf("H_core:\n"); my_print_matrix(H_core);
	printf("S^-1/2:\n"); my_print_matrix(S_invsqrt);
#endif

	// matrices, vector and variables to be used in SCF
	gsl_matrix *D_prev = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_matrix *G      = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_matrix *Fock   = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_matrix *Coef   = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_matrix *D      = gsl_matrix_alloc(p_basis->num, p_basis->num);
	gsl_vector *emo    = gsl_vector_alloc(p_basis->num);
	double ene_elec, ene_total, ene_prev;

	// initialize density matrix
	gsl_matrix_set_zero(D_prev);
	gsl_matrix_set_zero(D);
	ene_prev = 0.0;


	// Generalized Wolfsberg-Helmholtz initial guess
	init_guess_GWH(p_basis, H_core, S, Fock);
	Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
	Coef_to_Dens(p_basis->num, n_occ, Coef, D_prev);


	// DIIS error and Fock matrices
	double ***diis_err  = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
	double ***diis_Fock = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
	int idiis;
	for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
	{
		diis_err[idiis]  = (double **)my_malloc(sizeof(double *) * p_basis->num);
		diis_Fock[idiis] = (double **)my_malloc(sizeof(double *) * p_basis->num);
		for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
		{
			diis_err[idiis][ibasis]  = (double *)my_malloc(sizeof(double) * p_basis->num);
			diis_Fock[idiis][ibasis] = (double *)my_malloc(sizeof(double) * p_basis->num);
		}
	}

	// DIIS index and dimension
	int diis_index = 0;
	int diis_dim = 0;
	double delta_DIIS;

	fprintf(stdout, "%5s %20s %20s %20s %20s\n",
			"Iter", "E_total", "delta_E", "rms_D", "delta_DIIS");


	/*
	// Q: sqrt(ab|ab) for prescreening of two-electron integrals
	gsl_matrix *Q = gsl_matrix_alloc(p_basis->num, p_basis->num);
	form_Q(p_basis, Q);
	*/


	// start SCF iterations
	int iter = 0;
	while (1)
	{
		// SCF procedure:
		// Form new Fock matrix
		// F' = S^-1/2 * F * S^-1/2
		// diagonalize F' matrix to get C'
		// C = S^-1/2 * C'
		// compute new density matrix

		//form_G(p_basis->num, D_prev, ERI, G);
		//direct_form_G(p_basis, D_prev, Q, G);

		// use GPU-calculated two-electron integrals
		form_G(p_basis->num, D_prev, h_eri, G);

#ifdef DEBUG
		printf("G:\n"); my_print_matrix(G);
#endif

		form_Fock(p_basis->num, H_core, G, Fock);

		// DIIS
		if (iter > 0)
		{
			update_Fock_DIIS(&diis_dim, &diis_index, &delta_DIIS, 
						Fock, D_prev, S, p_basis, diis_err, diis_Fock);
		}

		// update density matrix and energies
		Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
		Coef_to_Dens(p_basis->num, n_occ, Coef, D);

		ene_elec = get_elec_ene(p_basis->num, D, H_core, Fock);
		ene_total = ene_nucl + ene_elec;

#ifdef DEBUG
		printf("F:\n"); my_print_matrix(Fock);
		printf("C:\n"); my_print_matrix(Coef);
		printf("P:\n"); my_print_matrix(D);
#endif

		// check convergence
		double delta_E = ene_total - ene_prev;

		double rms_D = 0.0;
		int mu, nu;
		for (mu = 0; mu < p_basis->num; ++ mu)
		{
			for (nu = 0; nu < p_basis->num; ++ nu)
			{
				double dd = gsl_matrix_get(D, mu, nu) - 
							gsl_matrix_get(D_prev, mu, nu);
				rms_D += dd * dd;
			}
		}
		rms_D = sqrt(rms_D);

		fprintf(stdout, "%5d %20.10f", iter, ene_total);
		if (iter > 0) { fprintf(stdout, " %20.10f %20.10f", delta_E, rms_D); }
		if (iter > 1) { fprintf(stdout, " %20.10f", delta_DIIS); }
		fprintf(stdout, "\n");

		// convergence criteria
		if (fabs(delta_E) < 1.0e-10 &&
			rms_D < 1.0e-8 && delta_DIIS < 1.0e-8) { break; }

		// update energy and density matrix for the next iteration
		ene_prev = ene_total;
		gsl_matrix_memcpy(D_prev, D);

		// count iterations
		++ iter;
	}

	// SCF converged
	fprintf(stdout, "SCF converged! E_total = %20.10f\n", ene_total);

	// print MO information
	fprintf(stdout, "%5s %10s %15s %12s\n", "MO", "State", "E(Eh)", "E(eV)");
	for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
	{
		char occ[10];
		if (ibasis < n_occ) { strcpy(occ, "occ."); }
		else { strcpy(occ, "virt."); }

		double ener = gsl_vector_get(emo, ibasis);
		fprintf(stdout, "%5d %10s %15.5f %12.2f\n",
				ibasis + 1, occ, ener, ener * HARTREE2EV);
	}


	//====== free allocated memories ========

	// free DIIS error and Fock matrices
	for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
	{
		for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
		{
			free(diis_err[idiis][ibasis]);
			free(diis_Fock[idiis][ibasis]);
		}
		free(diis_err[idiis]);
		free(diis_Fock[idiis]);
	}
	free(diis_err);
	free(diis_Fock);

	// free arrays for one- and two-electron integral
	gsl_matrix_free(S);
	gsl_matrix_free(T);
	gsl_matrix_free(V);
	//free(ERI);

	//gsl_matrix_free(Q);

	// free matrices and vector for SCF
	gsl_matrix_free(H_core);
	gsl_matrix_free(S_invsqrt);
	gsl_matrix_free(D_prev);
	gsl_matrix_free(G);
	gsl_matrix_free(Fock);
	gsl_matrix_free(Coef);
	gsl_matrix_free(D);
	gsl_vector_free(emo);

	// free arrays for geometry
	for (iatom = 0; iatom < p_atom->num; ++ iatom)
	{
		free(p_atom->pos[iatom]);
		free(p_atom->name[iatom]);
	}
	free(p_atom->pos);
	free(p_atom->name);

	free(p_atom->nuc_chg);

	// free arrays for basis set
	for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
	{
		free(p_basis->expon[ibasis]);
		free(p_basis->coef[ibasis]);
		free(p_basis->xbas[ibasis]);
		free(p_basis->lmn[ibasis]);
		free(p_basis->norm[ibasis]);
	}
	free(p_basis->expon);
	free(p_basis->coef);
	free(p_basis->xbas);
	free(p_basis->lmn);
	free(p_basis->norm);

	free(p_basis->nprims);


	//====== the end of program ========

	return 0;
}
