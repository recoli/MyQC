#include "hip/hip_runtime.h"
/*****************************************************************************
 This file is part of the XLQC program.                                      
 Copyright (C) 2015 Xin Li <lixin.reco@gmail.com>                            
                                                                           
 Filename:  main.cu                                                      
 License:   BSD 3-Clause License

 This software is provided by the copyright holders and contributors "as is"
 and any express or implied warranties, including, but not limited to, the
 implied warranties of merchantability and fitness for a particular purpose are
 disclaimed. In no event shall the copyright holder or contributors be liable
 for any direct, indirect, incidental, special, exemplary, or consequential
 damages (including, but not limited to, procurement of substitute goods or
 services; loss of use, data, or profits; or business interruption) however
 caused and on any theory of liability, whether in contract, strict liability,
 or tort (including negligence or otherwise) arising in any way out of the use
 of this software, even if advised of the possibility of such damage.
 *****************************************************************************/

#include <ctime>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <string>
#include <iostream>

#include <gsl/gsl_math.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_eigen.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_linalg.h>

#include "int_lib/cints.h"
#include "int_lib/crys.h"
#include "int_lib/chgp.h"

#include "typedef.h"
#include "basis.h"
#include "scf.h"

#include "cuda_rys_sp.h"
//#include "cuda_rys_dp.h"

int main(int argc, char* argv[])
{
    hipFree(0);

    // initialize timer
    clock_t t0, t1;
    double  time_in_sec, time_total;
    double  time_mat_J;

    t0 = clock();
    std::string time_txt ("");
    time_total = 0.0;
    time_mat_J = 0.0;

    // use spherical harmonic d function?
    const int use_5d = 1;
    // use double precision?
    //const int use_dp = 0;

    Atom   *p_atom   = (Atom *)my_malloc(sizeof(Atom) * 1);
    Basis  *p_basis  = (Basis *)my_malloc(sizeof(Basis) * 1);

    //====== parse geom.dat ========

    // get number of atoms
    p_atom->num = get_natoms();
    fprintf(stdout, "Natoms = %d\n", p_atom->num);

    // atomic coordinates and atom name
    p_atom->pos = (double **)my_malloc(sizeof(double *) * p_atom->num);
    p_atom->name = (char **)my_malloc(sizeof(char *) * p_atom->num);

    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        p_atom->pos[iatom] = (double *)my_malloc(sizeof(double) * CART_DIM);
        p_atom->name[iatom] = (char *)my_malloc(sizeof(char) * 5);
    }

    // nuclear charge
    p_atom->nuc_chg = (int *)my_malloc(sizeof(int) * p_atom->num);

    // read atomic positions, nuclear charge and atom name
    read_geom(p_atom);

    fprintf(stdout, "Coordinates in atomic unit:\n");
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        fprintf(stdout, "%s (%.1f)  %.10f  %.10f  %.10f\n", 
                p_atom->name[iatom], (double)p_atom->nuc_chg[iatom],
                p_atom->pos[iatom][0], p_atom->pos[iatom][1], p_atom->pos[iatom][2]);
    }
    
    // nuclear repulsion energy
    double ene_nucl = calc_ene_nucl(p_atom);
    fprintf(stdout, "Nuclear repulsion = %-20.10f\n", ene_nucl);


    //====== parse basis.dat ========

    // parse basis functions
    parse_basis(p_atom, p_basis, use_5d);

    fprintf(stdout, "System Nbasis = %d\n", p_basis->num);

    // basis function exponents, coefficients, and normalization factors
    p_basis->expon = (double **)my_malloc(sizeof(double *) * p_basis->num);
    p_basis->coef  = (double **)my_malloc(sizeof(double *) * p_basis->num);
    p_basis->norm  = (double **)my_malloc(sizeof(double *) * p_basis->num);

    // number of primitive functions in each contracted funciton
    p_basis->nprims = (int *)my_malloc(sizeof(int) * p_basis->num);

    // Cartesian coordinates and l,m,n numbers
    p_basis->xbas  = (double *)my_malloc(sizeof(double) * p_basis->num);
    p_basis->ybas  = (double *)my_malloc(sizeof(double) * p_basis->num);
    p_basis->zbas  = (double *)my_malloc(sizeof(double) * p_basis->num);

    p_basis->lx = (int **)my_malloc(sizeof(int *) * p_basis->num);
    p_basis->ly = (int **)my_malloc(sizeof(int *) * p_basis->num);
    p_basis->lz = (int **)my_malloc(sizeof(int *) * p_basis->num);

    // read basis set (also calculate normalization factors)
    read_basis(p_atom, p_basis, use_5d);

#ifdef DEBUG
    print_basis(p_basis);
#endif

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_Basis    = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    //====== one- and two-electron integrals ========

    // overlap, kinetic energy and nuclear attraction integral
    gsl_matrix *S = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *T = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *V = gsl_matrix_alloc(p_basis->num, p_basis->num);

    // two-electron ingetral
    int n_combi = p_basis->num * (p_basis->num + 1) / 2;

    for (int a = 0; a < p_basis->num; ++ a)
    {
        for (int b = 0; b <= a; ++ b)
        {
            // overlap
            double s = calc_int_overlap(p_basis, a, b);

            // kinetic energy
            double t = calc_int_kinetic(p_basis, a, b);

            // nuclear repulsion
            double v = calc_int_nuc_attr(p_basis, a, b, p_atom);

            // save one-electron integrals in matrices
            gsl_matrix_set(S, a, b, s);
            gsl_matrix_set(T, a, b, t);
            gsl_matrix_set(V, a, b, v);
            if (a != b)
            {
                gsl_matrix_set(S, b, a, s);
                gsl_matrix_set(T, b, a, t);
                gsl_matrix_set(V, b, a, v);
            }
        }
    }

    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_1e_Ints  = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // number of primitive bf
    int n_prim_basis = 0;
    for (int a = 0; a < p_basis->num; ++ a) 
    {
        n_prim_basis += p_basis->nprims[a];
    }

    size_t n_PF_bytes_int  = sizeof(int) * n_prim_basis;
    size_t n_PF2_bytes_int = sizeof(int) * n_prim_basis * n_prim_basis;

    // idx_PI: an array of dimension n_prim_basis x n_prim_basis
    // returns the index of bra/ket for primitive integrals
    // returns -1 if the combination is not considered
    int *h_idx_PI = (int *)my_malloc(n_PF2_bytes_int);
    for (int i = 0; i < n_prim_basis; ++ i) 
    {
        for (int k = 0; k < n_prim_basis; ++ k) 
        {
            h_idx_PI[i * n_prim_basis + k] = -1;
        }
    }

    // number of bra/ket pairs for primitive integrals
    int n_prim_combi = 0;
    for (int a = 0; a < p_basis->num; ++ a)
    {
        int lena = p_basis->nprims[a];

        int count_prim_a = 0;
        for (int tmp = 0; tmp < a; ++ tmp)
        {
            count_prim_a += p_basis->nprims[tmp];
        }

        for (int b = 0; b <= a; ++ b)
        {
            int lenb = p_basis->nprims[b];

            int count_prim_b = 0;
            for (int tmp = 0; tmp < b; ++ tmp)
            {
                count_prim_b += p_basis->nprims[tmp];
            }
        
            for (int i = 0; i < lena; ++ i)
            {
                int ai = count_prim_a + i;
                for (int j = 0; j < lenb; ++ j)
                {
                    int bj = count_prim_b + j;

                    // update idx_PI for bra/ket pairs
                    h_idx_PI[ai * n_prim_basis + bj] = n_prim_combi;

                    // update number of bra/ket pairs for PI
                    ++ n_prim_combi;
                }
            }
        }
    }


    // idx_CF: an array of dimension n_prim_basis
    // returns the index of contracted bf for a primitive bf
    int *h_idx_CF = (int *)my_malloc(n_PF_bytes_int);
    for (int a = 0; a < p_basis->num; ++ a)
    {
        int lena = p_basis->nprims[a];

        int count_prim_a = 0;
        for (int tmp = 0; tmp < a; ++ tmp)
        {
            count_prim_a += p_basis->nprims[tmp];
        }

        for (int i = 0; i < lena; ++ i)
        {
            h_idx_CF[count_prim_a + i] = a;
        }
    }


    // allocate memory for arrays on host
    // CI:  contracted integrals
    // PI:  primitive integrals
    size_t n_CI_bytes     = sizeof(double) * n_combi;
    size_t n_CI_bytes_int = sizeof(int)    * n_combi;
    size_t n_PI_bytes     = sizeof(double) * n_prim_combi;
    size_t n_PI_bytes_int = sizeof(int)    * n_prim_combi;


    // idx_CI: returns the index of CI pair for a PI pair
    int *h_idx_CI = (int *)my_malloc(n_PI_bytes_int);
    int i_prim_combi = 0;
    for (int a = 0; a < p_basis->num; ++ a)
    {
        for (int b = 0; b <= a; ++ b)
        {
            for (int i = 0; i < p_basis->nprims[a]; ++ i)
            {
                for (int j = 0; j < p_basis->nprims[b]; ++ j)
                {
                    // update idx_CI for bra/ket pairs
                    h_idx_CI[i_prim_combi] = ij2intindex(a,b);

                    // update number of bra/ket pairs for PI
                    ++ i_prim_combi;
                }
            }
        }
    }


    double *h_xa = (double *)my_malloc(n_CI_bytes);
    double *h_ya = (double *)my_malloc(n_CI_bytes);
    double *h_za = (double *)my_malloc(n_CI_bytes);
    double *h_xb = (double *)my_malloc(n_CI_bytes);
    double *h_yb = (double *)my_malloc(n_CI_bytes);
    double *h_zb = (double *)my_malloc(n_CI_bytes);

    int *h_la = (int *)my_malloc(n_PI_bytes_int);
    int *h_ma = (int *)my_malloc(n_PI_bytes_int);
    int *h_na = (int *)my_malloc(n_PI_bytes_int);
    int *h_lb = (int *)my_malloc(n_PI_bytes_int);
    int *h_mb = (int *)my_malloc(n_PI_bytes_int);
    int *h_nb = (int *)my_malloc(n_PI_bytes_int);

    // note that 'anorm' is absorbed into 'acoef'
    double *h_aexps = (double *)my_malloc(n_PI_bytes);
    double *h_acoef = (double *)my_malloc(n_PI_bytes);
    // note that 'bnorm' is absorbed into 'bcoef'
    double *h_bexps = (double *)my_malloc(n_PI_bytes);
    double *h_bcoef = (double *)my_malloc(n_PI_bytes);

    int *h_start_contr = (int *)my_malloc(n_CI_bytes_int);
    int *h_end_contr   = (int *)my_malloc(n_CI_bytes_int);

    // D: density matrix
    // J: Coulomb matrix
    // K: exchange matrix
    // Q: Schwartz pre-screening matrix
    double *h_mat_D = (double *)my_malloc(n_CI_bytes);
    double *h_mat_J = (double *)my_malloc(n_CI_bytes);
    double *h_mat_K = (double *)my_malloc(n_CI_bytes);
    double *h_mat_Q = (double *)my_malloc(n_CI_bytes);

    // J_PI and K_PI: for 1T1PI computation on GPUs
    double *h_mat_J_PI = (double *)my_malloc(n_PI_bytes);
    double *h_mat_K_PI = (double *)my_malloc(n_PI_bytes);


    // fill arrays on host
    // index_prim counts primitive integrals
    // index_contr counts contracted integrals
    int index_prim = 0;
    int index_contr = 0;

    for (int a = 0; a < p_basis->num; ++ a)
    {
        int lena = p_basis->nprims[a];
        for (int b = 0; b <= a; ++ b)
        {
            int lenb = p_basis->nprims[b];

            h_start_contr[index_contr] = index_prim;

            h_xa[index_contr] = p_basis->xbas[a];
            h_ya[index_contr] = p_basis->ybas[a];
            h_za[index_contr] = p_basis->zbas[a];

            h_xb[index_contr] = p_basis->xbas[b];
            h_yb[index_contr] = p_basis->ybas[b];
            h_zb[index_contr] = p_basis->zbas[b];
                            
            for (int i = 0; i < lena; ++ i)
            {
                for (int j = 0; j < lenb; ++ j)
                {
                    // note that 'anorm' is absorbed into 'acoef'
                    h_aexps[index_prim] = p_basis->expon[a][i];
                    h_acoef[index_prim] = p_basis->coef[a][i] * p_basis->norm[a][i];

                    // note that 'bnorm' is absorbed into 'bcoef'
                    h_bexps[index_prim] = p_basis->expon[b][j];
                    h_bcoef[index_prim] = p_basis->coef[b][j] * p_basis->norm[b][j];

                    h_la[index_prim] = p_basis->lx[a][i];
                    h_ma[index_prim] = p_basis->ly[a][i];
                    h_na[index_prim] = p_basis->lz[a][i];

                    h_lb[index_prim] = p_basis->lx[b][j];
                    h_mb[index_prim] = p_basis->ly[b][j];
                    h_nb[index_prim] = p_basis->lz[b][j];

                    ++ index_prim;
                }
            }

            h_end_contr[index_contr] = index_prim - 1;

            ++ index_contr;
        }
    }
    fprintf(stdout, "Num_Prim_Combi  = %d (%d)\n", index_prim, n_prim_combi);
    fprintf(stdout, "Num_Contr_Combi = %d (%d)\n", index_contr, n_combi);

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_2e_Prep  = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // initialize arrays on device
    double *dev_xa, *dev_ya, *dev_za;
    double *dev_xb, *dev_yb, *dev_zb;
    int    *dev_la, *dev_ma, *dev_na;
    int    *dev_lb, *dev_mb, *dev_nb;
    double *dev_aexps, *dev_acoef;
    double *dev_bexps, *dev_bcoef;

    int *dev_start_contr, *dev_end_contr;

    double *dev_mat_D, *dev_mat_Q, *dev_mat_J_PI, *dev_mat_K_PI;

    int *dev_idx_CI, *dev_idx_PI, *dev_idx_CF;

    // allocate memories for arrays on device
    /*
    fprintf(stdout, "Mem_on_Device = %zu MB\n",
            (n_CI_bytes*9 + n_PI_bytes_int*6 + n_PI_bytes*1 + n_CI_bytes_int*2) / 1000000);
    */

    my_cuda_safe(hipMalloc((void**)&dev_xa, n_CI_bytes),"alloc_xa");
    my_cuda_safe(hipMalloc((void**)&dev_ya, n_CI_bytes),"alloc_ya");
    my_cuda_safe(hipMalloc((void**)&dev_za, n_CI_bytes),"alloc_za");
    my_cuda_safe(hipMalloc((void**)&dev_xb, n_CI_bytes),"alloc_xb");
    my_cuda_safe(hipMalloc((void**)&dev_yb, n_CI_bytes),"alloc_yb");
    my_cuda_safe(hipMalloc((void**)&dev_zb, n_CI_bytes),"alloc_zb");

    my_cuda_safe(hipMalloc((void**)&dev_la, n_PI_bytes_int),"alloc_la");
    my_cuda_safe(hipMalloc((void**)&dev_ma, n_PI_bytes_int),"alloc_ma");
    my_cuda_safe(hipMalloc((void**)&dev_na, n_PI_bytes_int),"alloc_na");
    my_cuda_safe(hipMalloc((void**)&dev_lb, n_PI_bytes_int),"alloc_lb");
    my_cuda_safe(hipMalloc((void**)&dev_mb, n_PI_bytes_int),"alloc_mb");
    my_cuda_safe(hipMalloc((void**)&dev_nb, n_PI_bytes_int),"alloc_nb");

    my_cuda_safe(hipMalloc((void**)&dev_aexps, n_PI_bytes),"alloc_aexps");
    my_cuda_safe(hipMalloc((void**)&dev_acoef, n_PI_bytes),"alloc_acoef");
    my_cuda_safe(hipMalloc((void**)&dev_bexps, n_PI_bytes),"alloc_bexps");
    my_cuda_safe(hipMalloc((void**)&dev_bcoef, n_PI_bytes),"alloc_bcoef");

    my_cuda_safe(hipMalloc((void**)&dev_start_contr, n_CI_bytes_int),"alloc_st");
    my_cuda_safe(hipMalloc((void**)&dev_end_contr,   n_CI_bytes_int),"alloc_ed");

    my_cuda_safe(hipMalloc((void**)&dev_mat_D, n_CI_bytes),"alloc_D");
    my_cuda_safe(hipMalloc((void**)&dev_mat_Q, n_CI_bytes),"alloc_Q");

    my_cuda_safe(hipMalloc((void**)&dev_mat_J_PI, n_PI_bytes),"alloc_J_PI");
    my_cuda_safe(hipMalloc((void**)&dev_mat_K_PI, n_PI_bytes),"alloc_K_PI");

    my_cuda_safe(hipMalloc((void**)&dev_idx_PI, n_PF2_bytes_int),"alloc_idxPI");
    my_cuda_safe(hipMalloc((void**)&dev_idx_CF, n_PF_bytes_int), "alloc_idxCF");

    my_cuda_safe(hipMalloc((void**)&dev_idx_CI, n_PI_bytes_int),"alloc_ed");


    // copy data from host to device
    my_cuda_safe(hipMemcpy(dev_xa, h_xa, n_CI_bytes, hipMemcpyHostToDevice),"mem_xa");
    my_cuda_safe(hipMemcpy(dev_ya, h_ya, n_CI_bytes, hipMemcpyHostToDevice),"mem_ya");
    my_cuda_safe(hipMemcpy(dev_za, h_za, n_CI_bytes, hipMemcpyHostToDevice),"mem_za");
    my_cuda_safe(hipMemcpy(dev_xb, h_xb, n_CI_bytes, hipMemcpyHostToDevice),"mem_xb");
    my_cuda_safe(hipMemcpy(dev_yb, h_yb, n_CI_bytes, hipMemcpyHostToDevice),"mem_yb");
    my_cuda_safe(hipMemcpy(dev_zb, h_zb, n_CI_bytes, hipMemcpyHostToDevice),"mem_zb");

    my_cuda_safe(hipMemcpy(dev_la, h_la, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_la");
    my_cuda_safe(hipMemcpy(dev_ma, h_ma, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_ma");
    my_cuda_safe(hipMemcpy(dev_na, h_na, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_na");
    my_cuda_safe(hipMemcpy(dev_lb, h_lb, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_lb");
    my_cuda_safe(hipMemcpy(dev_mb, h_mb, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_mb");
    my_cuda_safe(hipMemcpy(dev_nb, h_nb, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_nb");

    my_cuda_safe(hipMemcpy(dev_aexps, h_aexps, n_PI_bytes, hipMemcpyHostToDevice),"mem_ae");
    my_cuda_safe(hipMemcpy(dev_acoef, h_acoef, n_PI_bytes, hipMemcpyHostToDevice),"mem_ac");
    my_cuda_safe(hipMemcpy(dev_bexps, h_bexps, n_PI_bytes, hipMemcpyHostToDevice),"mem_be");
    my_cuda_safe(hipMemcpy(dev_bcoef, h_bcoef, n_PI_bytes, hipMemcpyHostToDevice),"mem_bc");

    my_cuda_safe(hipMemcpy(dev_start_contr, h_start_contr, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_start");
    my_cuda_safe(hipMemcpy(dev_end_contr,   h_end_contr,   n_CI_bytes_int, hipMemcpyHostToDevice),"mem_end");

    my_cuda_safe(hipMemcpy(dev_idx_PI, h_idx_PI, n_PF2_bytes_int, hipMemcpyHostToDevice),"mem_idxPI");
    my_cuda_safe(hipMemcpy(dev_idx_CF, h_idx_CF, n_PF_bytes_int,  hipMemcpyHostToDevice),"mem_idxCF");

    my_cuda_safe(hipMemcpy(dev_idx_CI, h_idx_CI, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_idxCI");


    // create 8x8 thread blocks
    dim3 block_size;
    block_size.x = BLOCKSIZE;
    block_size.y = BLOCKSIZE;

    // configure a two dimensional grid as well
    dim3 grid_size;
    //grid_size.x = n_combi / block_size.x + (n_combi % block_size.x ? 1 : 0);
    //grid_size.y = n_combi / block_size.y + (n_combi % block_size.y ? 1 : 0);


    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    //time_txt += "Time_2e_GPU   = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    //====== start SCF calculation ========

    // NOTE: assume zero charge and closed-shell electronics structure
    int n_elec = 0;
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        n_elec += p_atom->nuc_chg[iatom];
    }

    if (n_elec % 2 != 0)
    {
        fprintf(stderr, "Error: Number of electrons (%d) is not even!\n", n_elec);
    }

    int n_occ = n_elec / 2;


    // get core Hamiltonian
    gsl_matrix *H_core = gsl_matrix_alloc(p_basis->num, p_basis->num);
    sum_H_core(p_basis->num, H_core, T, V);

    // get S^-1/2
    gsl_matrix *S_invsqrt = gsl_matrix_alloc(p_basis->num, p_basis->num);
    diag_overlap(p_basis->num, S, S_invsqrt);

#ifdef DEBUG
    printf("S:\n"); my_print_matrix(S);
    printf("T:\n"); my_print_matrix(T);
    printf("V:\n"); my_print_matrix(V);
    printf("H_core:\n"); my_print_matrix(H_core);
    printf("S^-1/2:\n"); my_print_matrix(S_invsqrt);
#endif

    // matrices, vector and variables to be used in SCF
    gsl_matrix *D_prev = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *Fock   = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *Coef   = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *D      = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_vector *emo    = gsl_vector_alloc(p_basis->num);
    double ene_elec, ene_total, ene_prev;

    // Coulomb(J) and exchange(K) matrices
    gsl_matrix *J = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *K = gsl_matrix_alloc(p_basis->num, p_basis->num);

    // initialize density matrix
    gsl_matrix_set_zero(D_prev);
    gsl_matrix_set_zero(D);
    ene_prev = 0.0;


    // Generalized Wolfsberg-Helmholtz initial guess
    init_guess_GWH(p_basis, H_core, S, Fock);
    Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
    Coef_to_Dens(p_basis->num, n_occ, Coef, D_prev);


    // DIIS error and Fock matrices
    double ***diis_err  = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
    double ***diis_Fock = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
    int idiis, ibasis;
    for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
    {
        diis_err[idiis]  = (double **)my_malloc(sizeof(double *) * p_basis->num);
        diis_Fock[idiis] = (double **)my_malloc(sizeof(double *) * p_basis->num);
        for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
        {
            diis_err[idiis][ibasis]  = (double *)my_malloc(sizeof(double) * p_basis->num);
            diis_Fock[idiis][ibasis] = (double *)my_malloc(sizeof(double) * p_basis->num);
        }
    }

    // DIIS index and dimension
    int diis_index = 0;
    int diis_dim = 0;
    double delta_DIIS;

    fprintf(stdout, "%5s %20s %20s %20s %20s\n",
            "Iter", "E_total", "delta_E", "rms_D", "delta_DIIS");


    // mat_Q: sqrt(ab|ab) for prescreening of two-electron integrals
    for (int a = 0; a < p_basis->num; ++ a) {
        for (int b = 0; b <= a; ++ b) {
            h_mat_Q[ij2intindex(a,b)] = calc_int_eri_rys(p_basis, a, b, a, b);
        }
    }

    my_cuda_safe(hipMemcpy(dev_mat_Q, h_mat_Q, n_CI_bytes, hipMemcpyHostToDevice),"mem_Q");

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_SCF_Init = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // start SCF iterations
    int iter = 0;
    while (1)
    {
        // SCF procedure:
        // Form new Fock matrix
        // F' = S^-1/2 * F * S^-1/2
        // diagonalize F' matrix to get C'
        // C = S^-1/2 * C'
        // compute new density matrix


        // timer for J-matrix
        clock_t t2,t3;
        t2 = clock();


        // NOTE: h_mat_D and dev_mat_D already contains the 2.0 factor for non-diagonal elements
        // This is convenient for J-matrix formation
        for (int a = 0; a < p_basis->num; ++ a) {
            for (int b = 0; b <= a; ++ b) {
                h_mat_D[ij2intindex(a,b)] = gsl_matrix_get(D_prev,a,b) * (a == b ? 1.0 : 2.0);
            }
        }

        my_cuda_safe(hipMemcpy(dev_mat_D, h_mat_D, n_CI_bytes, hipMemcpyHostToDevice),"mem_D");


        // use 1T1PI for J-matrix
        grid_size.x = n_prim_combi / block_size.x + (n_prim_combi % block_size.x ? 1 : 0);
        grid_size.y = 1;
    
        cuda_mat_J_PI<<<grid_size, block_size>>>
            (dev_xa,dev_ya,dev_za, dev_la,dev_ma,dev_na, dev_aexps,dev_acoef,
             dev_xb,dev_yb,dev_zb, dev_lb,dev_mb,dev_nb, dev_bexps,dev_bcoef,
             n_combi, n_prim_combi, dev_start_contr, dev_end_contr, dev_mat_D, dev_mat_J_PI, 
             dev_mat_Q, dev_idx_CI);

        my_cuda_safe(hipMemcpy(h_mat_J_PI, dev_mat_J_PI, n_PI_bytes, hipMemcpyDeviceToHost),"mem_J_PI");

        for (int idx_i = 0; idx_i < n_combi; ++ idx_i) 
        {
            h_mat_J[idx_i] = 0.0;
            int start_i = h_start_contr[idx_i];
            int end_i   = h_end_contr[idx_i];
            for (int i = start_i; i <= end_i; ++ i) 
            {
                h_mat_J[idx_i] += h_mat_J_PI[i];
            }
        }


        // use 1T1PI for K-matrix
        grid_size.x = n_prim_basis;
        grid_size.y = n_prim_basis;

        cuda_mat_K_PI<<<grid_size, block_size>>>
            (dev_xa,dev_ya,dev_za, dev_la,dev_ma,dev_na, dev_aexps,dev_acoef,
             dev_xb,dev_yb,dev_zb, dev_lb,dev_mb,dev_nb, dev_bexps,dev_bcoef,
             n_combi, n_prim_basis, dev_start_contr, dev_end_contr, dev_mat_D, dev_mat_K_PI, 
             dev_mat_Q, dev_idx_PI, dev_idx_CF, dev_idx_CI);

        my_cuda_safe(hipMemcpy(h_mat_K_PI, dev_mat_K_PI, n_PI_bytes, hipMemcpyDeviceToHost),"mem_K_PI");

        for (int idx_i = 0; idx_i < n_combi; ++ idx_i) 
        {
            h_mat_K[idx_i] = 0.0;
            int start_i = h_start_contr[idx_i];
            int end_i   = h_end_contr[idx_i];
            for (int i = start_i; i <= end_i; ++ i) 
            {
                h_mat_K[idx_i] += h_mat_K_PI[i];
            }
        }


        // use J and K matrix from GPU
        for (int a = 0; a < p_basis->num; ++ a) {
            for (int b = 0; b < p_basis->num; ++ b) {
                double Jab = h_mat_J[ij2intindex(a,b)];
                double Kab = h_mat_K[ij2intindex(a,b)];
                gsl_matrix_set(J,a,b,Jab);
                gsl_matrix_set(K,a,b,Kab);
            }
        }

        t3 = clock();
        time_in_sec = (t3 - t2) / (double)CLOCKS_PER_SEC;
        time_mat_J += time_in_sec;


#ifdef DEBUG
        printf("J:\n"); my_print_matrix(J);
        printf("K:\n"); my_print_matrix(K);
#endif

        form_Fock(p_basis->num, H_core, J, K, Fock);

        // DIIS
        if (iter > 0)
        {
            update_Fock_DIIS(&diis_dim, &diis_index, &delta_DIIS, 
                Fock, D_prev, S, p_basis, diis_err, diis_Fock);
        }

        // update density matrix and energies
        Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
        Coef_to_Dens(p_basis->num, n_occ, Coef, D);

        ene_elec = get_elec_ene(p_basis->num, D, H_core, Fock);
        ene_total = ene_nucl + ene_elec;

#ifdef DEBUG
        printf("F:\n"); my_print_matrix(Fock);
        printf("C:\n"); my_print_matrix(Coef);
        printf("P:\n"); my_print_matrix(D);
#endif

        // check convergence
        double delta_E = ene_total - ene_prev;

        double rms_D = 0.0;
        int mu, nu;
        for (mu = 0; mu < p_basis->num; ++ mu)
        {
            for (nu = 0; nu < p_basis->num; ++ nu)
            {
                double dd = gsl_matrix_get(D, mu, nu) - 
                            gsl_matrix_get(D_prev, mu, nu);
                rms_D += dd * dd;
            }
        }
        rms_D = sqrt(rms_D);

        fprintf(stdout, "%5d %20.10f", iter, ene_total);
        if (iter > 0) { fprintf(stdout, " %20.10f %20.10f", delta_E, rms_D); }
        if (iter > 1) { fprintf(stdout, " %20.10f", delta_DIIS); }
        fprintf(stdout, "\n");

        // convergence criteria
        if (fabs(delta_E) < 1.0e-10 &&
            rms_D < 1.0e-8 && delta_DIIS < 1.0e-8) { break; }

        // update energy and density matrix for the next iteration
        ene_prev = ene_total;
        gsl_matrix_memcpy(D_prev, D);

        // count iterations
        ++ iter;
    }

    // SCF converged
    fprintf(stdout, "SCF converged! E_total = %20.10f\n", ene_total);

    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_SCF_Conv = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // print MO information
    fprintf(stdout, "%5s %10s %15s %12s\n", "MO", "State", "E(Eh)", "E(eV)");
    for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
    {
        char occ[10];
        if (ibasis < n_occ) { strcpy(occ, "occ."); }
        else { strcpy(occ, "virt."); }

        double ener = gsl_vector_get(emo, ibasis);
        fprintf(stdout, "%5d %10s %15.5f %12.2f\n",
                ibasis + 1, occ, ener, ener * HARTREE2EV);
    }


    //====== free allocated memories ========

    // free DIIS error and Fock matrices
    for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
    {
        for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
        {
            free(diis_err[idiis][ibasis]);
            free(diis_Fock[idiis][ibasis]);
        }
        free(diis_err[idiis]);
        free(diis_Fock[idiis]);
    }
    free(diis_err);
    free(diis_Fock);

    // free arrays for one- and two-electron integral
    gsl_matrix_free(S);
    gsl_matrix_free(T);
    gsl_matrix_free(V);

    // free matrices and vector for SCF
    gsl_matrix_free(H_core);
    gsl_matrix_free(S_invsqrt);
    gsl_matrix_free(D_prev);
    gsl_matrix_free(Fock);
    gsl_matrix_free(Coef);
    gsl_matrix_free(D);
    gsl_vector_free(emo);

    gsl_matrix_free(J);
    gsl_matrix_free(K);

    // free arrays for geometry
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        free(p_atom->pos[iatom]);
        free(p_atom->name[iatom]);
    }
    free(p_atom->pos);
    free(p_atom->name);

    free(p_atom->nuc_chg);

    free(p_atom);

    // free arrays for basis set
    for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
    {
        free(p_basis->expon[ibasis]);
        free(p_basis->coef[ibasis]);
        free(p_basis->lx[ibasis]);
        free(p_basis->ly[ibasis]);
        free(p_basis->lz[ibasis]);
        free(p_basis->norm[ibasis]);
    }
    free(p_basis->expon);
    free(p_basis->coef);
    free(p_basis->lx);
    free(p_basis->ly);
    free(p_basis->lz);
    free(p_basis->norm);

    free(p_basis->xbas);
    free(p_basis->ybas);
    free(p_basis->zbas);

    free(p_basis->nprims);

    free(p_basis);

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_Finalize = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    std::cout << time_txt;
    std::cout << "Total time used " << time_total << " sec\n";

    std::cout << "MatJK time used " << time_mat_J << " sec\n";


    //====== the end of program ========

    return 0;
}
