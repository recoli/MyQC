#include "hip/hip_runtime.h"
/*****************************************************************************
 This file is part of the XLQC program.                                      
 Copyright (C) 2015 Xin Li <lixin.reco@gmail.com>                            
                                                                           
 Filename:  main.cu                                                      
 License:   BSD 3-Clause License

 This software is provided by the copyright holders and contributors "as is"
 and any express or implied warranties, including, but not limited to, the
 implied warranties of merchantability and fitness for a particular purpose are
 disclaimed. In no event shall the copyright holder or contributors be liable
 for any direct, indirect, incidental, special, exemplary, or consequential
 damages (including, but not limited to, procurement of substitute goods or
 services; loss of use, data, or profits; or business interruption) however
 caused and on any theory of liability, whether in contract, strict liability,
 or tort (including negligence or otherwise) arising in any way out of the use
 of this software, even if advised of the possibility of such damage.
 *****************************************************************************/

#include <ctime>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <string>
#include <iostream>

#include <gsl/gsl_math.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_eigen.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_linalg.h>

#include "int_lib/cints.h"
#include "int_lib/crys.h"
#include "int_lib/chgp.h"

#include "typedef.h"
#include "basis.h"
#include "scf.h"

#include "cuda_rys_sp.h"
#include "cuda_rys_dp.h"

int main(int argc, char* argv[])
{
    // initialize timer
    clock_t t0, t1;
    double  time_in_sec, time_total;

    t0 = clock();
    std::string time_txt ("");
    time_total = 0.0;

    // use spherical harmonic d function?
    const int use_5d = 1;
    // use double precision?
    const int use_dp = 0;

    Atom   *p_atom   = (Atom *)my_malloc(sizeof(Atom) * 1);
    Basis  *p_basis  = (Basis *)my_malloc(sizeof(Basis) * 1);

    //====== parse geom.dat ========

    // get number of atoms
    p_atom->num = get_natoms();
    fprintf(stdout, "Natoms = %d\n", p_atom->num);

    // atomic coordinates and atom name
    p_atom->pos = (double **)my_malloc(sizeof(double *) * p_atom->num);
    p_atom->name = (char **)my_malloc(sizeof(char *) * p_atom->num);

    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        p_atom->pos[iatom] = (double *)my_malloc(sizeof(double) * CART_DIM);
        p_atom->name[iatom] = (char *)my_malloc(sizeof(char) * 5);
    }

    // nuclear charge
    p_atom->nuc_chg = (int *)my_malloc(sizeof(int) * p_atom->num);

    // read atomic positions, nuclear charge and atom name
    read_geom(p_atom);

    fprintf(stdout, "Coordinates in atomic unit:\n");
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        fprintf(stdout, "%s (%.1f)  %.10f  %.10f  %.10f\n", 
                p_atom->name[iatom], (double)p_atom->nuc_chg[iatom],
                p_atom->pos[iatom][0], p_atom->pos[iatom][1], p_atom->pos[iatom][2]);
    }
    
    // nuclear repulsion energy
    double ene_nucl = calc_ene_nucl(p_atom);
    fprintf(stdout, "Nuclear repulsion = %-20.10f\n", ene_nucl);


    //====== parse basis.dat ========

    // parse basis functions
    parse_basis(p_atom, p_basis, use_5d);

    fprintf(stdout, "System Nbasis = %d\n", p_basis->num);

    // basis function exponents, coefficients, and normalization factors
    p_basis->expon = (double **)my_malloc(sizeof(double *) * p_basis->num);
    p_basis->coef  = (double **)my_malloc(sizeof(double *) * p_basis->num);
    p_basis->norm  = (double **)my_malloc(sizeof(double *) * p_basis->num);

    // number of primitive functions in each contracted funciton
    p_basis->nprims = (int *)my_malloc(sizeof(int) * p_basis->num);

    // Cartesian coordinates and l,m,n numbers
    p_basis->xbas  = (double *)my_malloc(sizeof(double) * p_basis->num);
    p_basis->ybas  = (double *)my_malloc(sizeof(double) * p_basis->num);
    p_basis->zbas  = (double *)my_malloc(sizeof(double) * p_basis->num);

    p_basis->lx = (int **)my_malloc(sizeof(int *) * p_basis->num);
    p_basis->ly = (int **)my_malloc(sizeof(int *) * p_basis->num);
    p_basis->lz = (int **)my_malloc(sizeof(int *) * p_basis->num);

    // read basis set (also calculate normalization factors)
    read_basis(p_atom, p_basis, use_5d);

#ifdef DEBUG
    print_basis(p_basis);
#endif

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_Basis    = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    //====== one- and two-electron integrals ========

    // overlap, kinetic energy and nuclear attraction integral
    gsl_matrix *S = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *T = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *V = gsl_matrix_alloc(p_basis->num, p_basis->num);

    // two-electron ingetral
    int n_combi = p_basis->num * (p_basis->num + 1) / 2;
    int n_eri = n_combi * (n_combi + 1) / 2;

    for (int a = 0; a < p_basis->num; ++ a)
    {
        for (int b = 0; b <= a; ++ b)
        {
            // overlap
            double s = calc_int_overlap(p_basis, a, b);

            // kinetic energy
            double t = calc_int_kinetic(p_basis, a, b);

            // nuclear repulsion
            double v = calc_int_nuc_attr(p_basis, a, b, p_atom);

            // save one-electron integrals in matrices
            gsl_matrix_set(S, a, b, s);
            gsl_matrix_set(T, a, b, t);
            gsl_matrix_set(V, a, b, v);
            if (a != b)
            {
                gsl_matrix_set(S, b, a, s);
                gsl_matrix_set(T, b, a, t);
                gsl_matrix_set(V, b, a, v);
            }
        }
    }

    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_1e_Ints  = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // count number of primitive integrals in a <bra| or |ket>
    int count_prim = 0;
    for (int a = 0; a < p_basis->num; ++ a)
    {
        int lena = p_basis->nprims[a];
        for (int b = 0; b <= a; ++ b)
        {
            int lenb = p_basis->nprims[b];
        
            for (int i = 0; i < lena; ++ i)
                for (int j = 0; j < lenb; ++ j)
                    ++ count_prim;
        }
    }

    // allocate memory for arrays on host
    // CI:  contracted integrals
    // PI:  primitive integrals
    // ERI: electron repulsion integrals
    size_t n_CI_bytes     = sizeof(double) * n_combi;
    size_t n_CI_bytes_int = sizeof(int)    * n_combi;
    size_t n_PI_bytes     = sizeof(double) * count_prim;
    size_t n_PI_bytes_int = sizeof(int)    * count_prim;
    size_t n_ERI_bytes    = sizeof(double) * n_eri;

    double *h_xa = (double *)my_malloc(n_CI_bytes);
    double *h_ya = (double *)my_malloc(n_CI_bytes);
    double *h_za = (double *)my_malloc(n_CI_bytes);
    double *h_xb = (double *)my_malloc(n_CI_bytes);
    double *h_yb = (double *)my_malloc(n_CI_bytes);
    double *h_zb = (double *)my_malloc(n_CI_bytes);

    int *h_la = (int *)my_malloc(n_PI_bytes_int);
    int *h_ma = (int *)my_malloc(n_PI_bytes_int);
    int *h_na = (int *)my_malloc(n_PI_bytes_int);
    int *h_lb = (int *)my_malloc(n_PI_bytes_int);
    int *h_mb = (int *)my_malloc(n_PI_bytes_int);
    int *h_nb = (int *)my_malloc(n_PI_bytes_int);

    // note that 'anorm' is absorbed into 'acoef'
    double *h_aexps = (double *)my_malloc(n_PI_bytes);
    double *h_acoef = (double *)my_malloc(n_PI_bytes);
    // note that 'bnorm' is absorbed into 'bcoef'
    double *h_bexps = (double *)my_malloc(n_PI_bytes);
    double *h_bcoef = (double *)my_malloc(n_PI_bytes);

    int *h_start_contr = (int *)my_malloc(n_CI_bytes_int);
    int *h_end_contr   = (int *)my_malloc(n_CI_bytes_int);

    double *h_eri = (double *)my_malloc(n_ERI_bytes);

    // fill arrays on host
    // index counts primitive integrals
    // index_contr counts contracted integrals
    int index = 0;
    int index_contr = 0;

    for (int a = 0; a < p_basis->num; ++ a)
    {
        int lena = p_basis->nprims[a];
        for (int b = 0; b <= a; ++ b)
        {
            int lenb = p_basis->nprims[b];

            h_start_contr[index_contr] = index;

            h_xa[index_contr] = p_basis->xbas[a];
            h_ya[index_contr] = p_basis->ybas[a];
            h_za[index_contr] = p_basis->zbas[a];

            h_xb[index_contr] = p_basis->xbas[b];
            h_yb[index_contr] = p_basis->ybas[b];
            h_zb[index_contr] = p_basis->zbas[b];
                            
            for (int i = 0; i < lena; ++ i)
            {
                for (int j = 0; j < lenb; ++ j)
                {
                    // note that 'anorm' is absorbed into 'acoef'
                    h_aexps[index] = p_basis->expon[a][i];
                    h_acoef[index] = p_basis->coef[a][i] * p_basis->norm[a][i];

                    // note that 'bnorm' is absorbed into 'bcoef'
                    h_bexps[index] = p_basis->expon[b][j];
                    h_bcoef[index] = p_basis->coef[b][j] * p_basis->norm[b][j];

                    h_la[index] = p_basis->lx[a][i];
                    h_ma[index] = p_basis->ly[a][i];
                    h_na[index] = p_basis->lz[a][i];

                    h_lb[index] = p_basis->lx[b][j];
                    h_mb[index] = p_basis->ly[b][j];
                    h_nb[index] = p_basis->lz[b][j];

                    ++ index;
                }
            }

            h_end_contr[index_contr] = index - 1;

            ++ index_contr;
        }
    }
    fprintf(stdout, "Num_Prim_Combi  = %d (%d)\n", index, count_prim);
    fprintf(stdout, "Num_Contr_Combi = %d (%d)\n", index_contr, n_combi);

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_2e_Prep  = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // initialize arrays on device
    double *dev_xa, *dev_ya, *dev_za;
    double *dev_xb, *dev_yb, *dev_zb;
    int    *dev_la, *dev_ma, *dev_na;
    int    *dev_lb, *dev_mb, *dev_nb;
    double *dev_aexps, *dev_acoef;
    double *dev_bexps, *dev_bcoef;

    dev_xa = NULL; dev_ya = NULL; dev_za = NULL;
    dev_xb = NULL; dev_yb = NULL; dev_zb = NULL;
    dev_la = NULL; dev_ma = NULL; dev_na = NULL;
    dev_lb = NULL; dev_mb = NULL; dev_nb = NULL;
    dev_aexps = NULL; dev_acoef = NULL;
    dev_bexps = NULL; dev_bcoef = NULL;

    int *dev_start_contr = NULL;
    int *dev_end_contr   = NULL;

    double *dev_eri = NULL;

    // allocate memories for arrays on device
    fprintf(stdout, "Mem_on_Device = %zu MB\n",
            (n_CI_bytes*6 + n_PI_bytes_int*6 + n_PI_bytes*4 + 
             n_CI_bytes_int*2 + n_ERI_bytes) / 1000000);

    hipMalloc((void**)&dev_xa, n_CI_bytes);
    hipMalloc((void**)&dev_ya, n_CI_bytes);
    hipMalloc((void**)&dev_za, n_CI_bytes);
    hipMalloc((void**)&dev_xb, n_CI_bytes);
    hipMalloc((void**)&dev_yb, n_CI_bytes);
    hipMalloc((void**)&dev_zb, n_CI_bytes);

    if(dev_xa == NULL || dev_ya == NULL || dev_za == NULL ||
       dev_xb == NULL || dev_yb == NULL || dev_zb == NULL)
    {
        fprintf(stderr, "Error: cannot hipMalloc for x_basis!\n");
        exit(1);
    }

    hipMalloc((void**)&dev_la, n_PI_bytes_int);
    hipMalloc((void**)&dev_ma, n_PI_bytes_int);
    hipMalloc((void**)&dev_na, n_PI_bytes_int);
    hipMalloc((void**)&dev_lb, n_PI_bytes_int);
    hipMalloc((void**)&dev_mb, n_PI_bytes_int);
    hipMalloc((void**)&dev_nb, n_PI_bytes_int);

    if(dev_la == NULL || dev_ma == NULL || dev_na == NULL ||
       dev_lb == NULL || dev_mb == NULL || dev_nb == NULL)
    {
        fprintf(stderr, "Error: cannot hipMalloc for l_basis!\n");
        exit(1);
    }

    hipMalloc((void**)&dev_aexps, n_PI_bytes);
    hipMalloc((void**)&dev_acoef, n_PI_bytes);
    hipMalloc((void**)&dev_bexps, n_PI_bytes);
    hipMalloc((void**)&dev_bcoef, n_PI_bytes);

    if(dev_aexps == NULL || dev_acoef == NULL ||
       dev_bexps == NULL || dev_bcoef == NULL)
    {
        fprintf(stderr, "Error: cannot hipMalloc for exps_basis!\n");
        exit(1);
    }

    hipMalloc((void**)&dev_start_contr, n_CI_bytes_int);
    hipMalloc((void**)&dev_end_contr,   n_CI_bytes_int);

    hipMalloc((void**)&dev_eri, n_ERI_bytes);

    if(dev_eri == NULL || dev_start_contr == NULL || dev_end_contr == NULL)
    {
        fprintf(stderr, "Error: cannot hipMalloc for dev_eri!\n");
        exit(1);
    }

    // copy data from host to device
    my_cuda_safe(hipMemcpy(dev_xa, h_xa, n_CI_bytes, hipMemcpyHostToDevice),"mem_xa");
    my_cuda_safe(hipMemcpy(dev_ya, h_ya, n_CI_bytes, hipMemcpyHostToDevice),"mem_ya");
    my_cuda_safe(hipMemcpy(dev_za, h_za, n_CI_bytes, hipMemcpyHostToDevice),"mem_za");
    my_cuda_safe(hipMemcpy(dev_xb, h_xb, n_CI_bytes, hipMemcpyHostToDevice),"mem_xb");
    my_cuda_safe(hipMemcpy(dev_yb, h_yb, n_CI_bytes, hipMemcpyHostToDevice),"mem_yb");
    my_cuda_safe(hipMemcpy(dev_zb, h_zb, n_CI_bytes, hipMemcpyHostToDevice),"mem_zb");

    my_cuda_safe(hipMemcpy(dev_la, h_la, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_la");
    my_cuda_safe(hipMemcpy(dev_ma, h_ma, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_ma");
    my_cuda_safe(hipMemcpy(dev_na, h_na, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_na");
    my_cuda_safe(hipMemcpy(dev_lb, h_lb, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_lb");
    my_cuda_safe(hipMemcpy(dev_mb, h_mb, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_mb");
    my_cuda_safe(hipMemcpy(dev_nb, h_nb, n_PI_bytes_int, hipMemcpyHostToDevice),"mem_nb");

    my_cuda_safe(hipMemcpy(dev_aexps, h_aexps, n_PI_bytes, hipMemcpyHostToDevice),"mem_ae");
    my_cuda_safe(hipMemcpy(dev_acoef, h_acoef, n_PI_bytes, hipMemcpyHostToDevice),"mem_ac");
    my_cuda_safe(hipMemcpy(dev_bexps, h_bexps, n_PI_bytes, hipMemcpyHostToDevice),"mem_be");
    my_cuda_safe(hipMemcpy(dev_bcoef, h_bcoef, n_PI_bytes, hipMemcpyHostToDevice),"mem_bc");

    my_cuda_safe(hipMemcpy(dev_start_contr, h_start_contr, n_CI_bytes_int, hipMemcpyHostToDevice),"mem_start");
    my_cuda_safe(hipMemcpy(dev_end_contr,   h_end_contr,   n_CI_bytes_int, hipMemcpyHostToDevice),"mem_end");

    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_2e_DMem  = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // create 8x8 thread blocks
    dim3 block_size;
    block_size.x = 8;
    block_size.y = 8;

    // configure a two dimensional grid as well
    dim3 grid_size;
    grid_size.x = n_combi / block_size.x + (n_combi % block_size.x ? 1 : 0);
    grid_size.y = n_combi / block_size.y + (n_combi % block_size.y ? 1 : 0);


    // launch the kernel to calculate two-electron integrals on GPU
    if (use_dp) {
        cuda_rys_eri_2d_dp<<<grid_size, block_size>>>
            (dev_xa,dev_ya,dev_za, dev_la,dev_ma,dev_na, dev_aexps,dev_acoef,
             dev_xb,dev_yb,dev_zb, dev_lb,dev_mb,dev_nb, dev_bexps,dev_bcoef,
             n_combi, dev_start_contr, dev_end_contr, dev_eri);
    } else {
        cuda_rys_eri_2d<<<grid_size, block_size>>>
            (dev_xa,dev_ya,dev_za, dev_la,dev_ma,dev_na, dev_aexps,dev_acoef,
             dev_xb,dev_yb,dev_zb, dev_lb,dev_mb,dev_nb, dev_bexps,dev_bcoef,
             n_combi, dev_start_contr, dev_end_contr, dev_eri);
    }

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_2e_GPU   = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // copy the results back to host
    my_cuda_safe(hipMemcpy(h_eri, dev_eri, n_ERI_bytes, hipMemcpyDeviceToHost),"mem_eri"); 

    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_2e_HMem  = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    //====== start SCF calculation ========

    // NOTE: assume zero charge and closed-shell electronics structure
    int n_elec = 0;
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        n_elec += p_atom->nuc_chg[iatom];
    }

    if (n_elec % 2 != 0)
    {
        fprintf(stderr, "Error: Number of electrons (%d) is not even!\n", n_elec);
    }

    int n_occ = n_elec / 2;


    // get core Hamiltonian
    gsl_matrix *H_core = gsl_matrix_alloc(p_basis->num, p_basis->num);
    sum_H_core(p_basis->num, H_core, T, V);

    // get S^-1/2
    gsl_matrix *S_invsqrt = gsl_matrix_alloc(p_basis->num, p_basis->num);
    diag_overlap(p_basis->num, S, S_invsqrt);

#ifdef DEBUG
    printf("S:\n"); my_print_matrix(S);
    printf("T:\n"); my_print_matrix(T);
    printf("V:\n"); my_print_matrix(V);
    printf("H_core:\n"); my_print_matrix(H_core);
    printf("S^-1/2:\n"); my_print_matrix(S_invsqrt);
#endif

    // matrices, vector and variables to be used in SCF
    gsl_matrix *D_prev = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *G      = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *Fock   = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *Coef   = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *D      = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_vector *emo    = gsl_vector_alloc(p_basis->num);
    double ene_elec, ene_total, ene_prev;

    // initialize density matrix
    gsl_matrix_set_zero(D_prev);
    gsl_matrix_set_zero(D);
    ene_prev = 0.0;


    // Generalized Wolfsberg-Helmholtz initial guess
    init_guess_GWH(p_basis, H_core, S, Fock);
    Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
    Coef_to_Dens(p_basis->num, n_occ, Coef, D_prev);


    // DIIS error and Fock matrices
    double ***diis_err  = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
    double ***diis_Fock = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
    int idiis, ibasis;
    for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
    {
        diis_err[idiis]  = (double **)my_malloc(sizeof(double *) * p_basis->num);
        diis_Fock[idiis] = (double **)my_malloc(sizeof(double *) * p_basis->num);
        for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
        {
            diis_err[idiis][ibasis]  = (double *)my_malloc(sizeof(double) * p_basis->num);
            diis_Fock[idiis][ibasis] = (double *)my_malloc(sizeof(double) * p_basis->num);
        }
    }

    // DIIS index and dimension
    int diis_index = 0;
    int diis_dim = 0;
    double delta_DIIS;

    fprintf(stdout, "%5s %20s %20s %20s %20s\n",
            "Iter", "E_total", "delta_E", "rms_D", "delta_DIIS");


    /*
    // Q: sqrt(ab|ab) for prescreening of two-electron integrals
    gsl_matrix *Q = gsl_matrix_alloc(p_basis->num, p_basis->num);
    form_Q(p_basis, Q);
    */

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_SCF_Init = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // start SCF iterations
    int iter = 0;
    while (1)
    {
        // SCF procedure:
        // Form new Fock matrix
        // F' = S^-1/2 * F * S^-1/2
        // diagonalize F' matrix to get C'
        // C = S^-1/2 * C'
        // compute new density matrix

        //form_G(p_basis->num, D_prev, ERI, G);
        //direct_form_G(p_basis, D_prev, Q, G);

        // use GPU-calculated two-electron integrals
        form_G(p_basis->num, D_prev, h_eri, G);

#ifdef DEBUG
        printf("G:\n"); my_print_matrix(G);
#endif

        form_Fock(p_basis->num, H_core, G, Fock);

        // DIIS
        if (iter > 0)
        {
            update_Fock_DIIS(&diis_dim, &diis_index, &delta_DIIS, 
                        Fock, D_prev, S, p_basis, diis_err, diis_Fock);
        }

        // update density matrix and energies
        Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
        Coef_to_Dens(p_basis->num, n_occ, Coef, D);

        ene_elec = get_elec_ene(p_basis->num, D, H_core, Fock);
        ene_total = ene_nucl + ene_elec;

#ifdef DEBUG
        printf("F:\n"); my_print_matrix(Fock);
        printf("C:\n"); my_print_matrix(Coef);
        printf("P:\n"); my_print_matrix(D);
#endif

        // check convergence
        double delta_E = ene_total - ene_prev;

        double rms_D = 0.0;
        int mu, nu;
        for (mu = 0; mu < p_basis->num; ++ mu)
        {
            for (nu = 0; nu < p_basis->num; ++ nu)
            {
                double dd = gsl_matrix_get(D, mu, nu) - 
                            gsl_matrix_get(D_prev, mu, nu);
                rms_D += dd * dd;
            }
        }
        rms_D = sqrt(rms_D);

        fprintf(stdout, "%5d %20.10f", iter, ene_total);
        if (iter > 0) { fprintf(stdout, " %20.10f %20.10f", delta_E, rms_D); }
        if (iter > 1) { fprintf(stdout, " %20.10f", delta_DIIS); }
        fprintf(stdout, "\n");

        // convergence criteria
        if (fabs(delta_E) < 1.0e-10 &&
            rms_D < 1.0e-8 && delta_DIIS < 1.0e-8) { break; }

        // update energy and density matrix for the next iteration
        ene_prev = ene_total;
        gsl_matrix_memcpy(D_prev, D);

        // count iterations
        ++ iter;
    }

    // SCF converged
    fprintf(stdout, "SCF converged! E_total = %20.10f\n", ene_total);

    t0 = clock();
    time_in_sec = (t0 - t1) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_SCF_Conv = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    // print MO information
    fprintf(stdout, "%5s %10s %15s %12s\n", "MO", "State", "E(Eh)", "E(eV)");
    for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
    {
        char occ[10];
        if (ibasis < n_occ) { strcpy(occ, "occ."); }
        else { strcpy(occ, "virt."); }

        double ener = gsl_vector_get(emo, ibasis);
        fprintf(stdout, "%5d %10s %15.5f %12.2f\n",
                ibasis + 1, occ, ener, ener * HARTREE2EV);
    }


    //====== free allocated memories ========

    // free DIIS error and Fock matrices
    for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
    {
        for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
        {
            free(diis_err[idiis][ibasis]);
            free(diis_Fock[idiis][ibasis]);
        }
        free(diis_err[idiis]);
        free(diis_Fock[idiis]);
    }
    free(diis_err);
    free(diis_Fock);

    // free arrays for one- and two-electron integral
    gsl_matrix_free(S);
    gsl_matrix_free(T);
    gsl_matrix_free(V);
    //free(ERI);

    //gsl_matrix_free(Q);

    // free matrices and vector for SCF
    gsl_matrix_free(H_core);
    gsl_matrix_free(S_invsqrt);
    gsl_matrix_free(D_prev);
    gsl_matrix_free(G);
    gsl_matrix_free(Fock);
    gsl_matrix_free(Coef);
    gsl_matrix_free(D);
    gsl_vector_free(emo);

    // free arrays for geometry
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        free(p_atom->pos[iatom]);
        free(p_atom->name[iatom]);
    }
    free(p_atom->pos);
    free(p_atom->name);

    free(p_atom->nuc_chg);

    free(p_atom);

    // free arrays for basis set
    for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
    {
        free(p_basis->expon[ibasis]);
        free(p_basis->coef[ibasis]);
        free(p_basis->lx[ibasis]);
        free(p_basis->ly[ibasis]);
        free(p_basis->lz[ibasis]);
        free(p_basis->norm[ibasis]);
    }
    free(p_basis->expon);
    free(p_basis->coef);
    free(p_basis->lx);
    free(p_basis->ly);
    free(p_basis->lz);
    free(p_basis->norm);

    free(p_basis->xbas);
    free(p_basis->ybas);
    free(p_basis->zbas);

    free(p_basis->nprims);

    free(p_basis);

    t1 = clock();
    time_in_sec = (t1 - t0) / (double)CLOCKS_PER_SEC;
    time_txt += "Time_Finalize = " + std::to_string(time_in_sec) + " sec\n";
    time_total += time_in_sec;


    std::cout << time_txt;
    std::cout << "Total time used " << time_total << " sec\n";


    //====== the end of program ========

    return 0;
}
